
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<algorithm>
#include<queue>
#include<vector>
#include<iostream>
using namespace std;

const double d = 0.85;
int V, E, L, M;

std::vector<std::vector<int>> in_edges;
std::vector<int> out_degree;


static const int blockSize = 1024;


__global__ void oneVertex(const int i, 
	const int V, 
	const double d,
	const int next,
	const int current, const int* flat_edges,
	const int* edge_starts,
	const int* arr_out_degree, double* arr_pr) {

	int idx = threadIdx.x;
	int sum = 0;


	for (int j = idx + edge_starts[i];
		j < edge_starts[i + 1]; j += blockSize) {
		int v = flat_edges[j];
		sum += arr_pr[v + current * V] / arr_out_degree[v];

	}

	__shared__ int r[blockSize];
	r[idx] = sum;
	__syncthreads();
	for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
		if (idx < size)
			r[idx] += r[idx + size];
		__syncthreads();
	}

	if (idx == 0) {
		arr_pr[i + next * V] = (1.0 - d) / V + d * r[0];
	}
	
	}


int main(int argc, char** argv) {
	FILE* fin = fopen(argv[1], "r");
	FILE* fout = fopen(argv[2], "w");
	fscanf(fin, "%d%d%d%d", &V, &E, &L, &M);
	in_edges.resize(V);
	out_degree = std::vector<int>(V, 0);

	int longest_in_edges = 0;
	int num_edges = 0;
	for (int i = 0; i < E; ++i) {
		int u, v;
		fscanf(fin, "%d%d", &u, &v);
		in_edges[v].push_back(u);
		++out_degree[u];

		// compute longest edge
		if (in_edges[v].size() > longest_in_edges) {
			longest_in_edges = in_edges[v].size();
		}

	}



	std::vector<double> pr[2];
	pr[0].resize(V);
	pr[1].resize(V);
	int current = 0;
	for (int i = 0; i < V; ++i) {
		pr[current][i] = 1.0 / V;
	}

	//create array equivalents 

	int* flat_edges = (int*)malloc(E * sizeof(int));
	int edge_starts[V + 1];

	int pos = 0; 

	for (int i = 0; i < V; ++i) {
		
		edge_starts[i] = pos; 

		for (int j = 0; j < in_edges[i].size(); j++) {
			flat_edges[pos] = in_edges[i][j];
			++pos;



		}
	}

	edge_starts[V] = E; 


	int** arr_in_edges = (int**)malloc(V * sizeof(int*));
	int arr_in_edges_count[V];

	for (int i = 0; i < V; ++i) {
		arr_in_edges[i] = (int*)malloc(in_edges[i].size() * sizeof(int));
		arr_in_edges_count[i] = in_edges[i].size();
		for (int j = 0; j < in_edges[i].size(); j++) {
			arr_in_edges[i][j] = in_edges[i][j];

		}
	}

	int arr_out_degree[V];
	for (int i = 0; i < V; ++i) {
		arr_out_degree[i] = out_degree[i];
	}

	double arr_pr[V*2];

	for (int i = 0; i < V; ++i) {
		arr_pr[i+current*V] = 1.0 / V;
	}



	////cuda allocate PR 
	int start = 0;
	int end = 0; 

	for (int iter = 0; iter < M; ++iter) {
		int next = 1 - current;
		for (int i = 0; i < V; ++i) {
			double sum = 0;

			start = edge_starts[i]; 
			end = edge_starts[i+1];

			for (int j = start; j < end; ++j) {
				int v = flat_edges[j];
				sum += arr_pr[v + current * V] / arr_out_degree[v];
			}

			arr_pr[i + next * V] = (1.0 - d) / V + d * sum;
		}
		current = next;
	}

	for (int i = 0; i < V; ++i) {
		pr[current][i] = arr_pr[i + current * V];
	}

	for (int i = 0; i < V; ++i) {
		fprintf(fout, "%.8f\n", pr[current][i]);
	}
	fclose(fin);
	fclose(fout);

	return 0;
}

