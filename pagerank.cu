#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<algorithm>
#include<queue>
#include<vector>
#include<iostream>
using namespace std;

const double d = 0.85;
int V, E, L, M;

std::vector<std::vector<int>> in_edges;
std::vector<int> out_degree;


static const int blockSize = 1024;
static const int blocks = 1024*64;


__global__ void sum_sections(
	const int V,
	const int total_edge_sections,
	const int current,
	const int* flat_edges,
	const int* cu_edge_sections,
	const int* cu_edge_sections_to_vertex,
	const int* arr_out_degree,
	const double* arr_pr,
	double* sections_result
) {

	int idx = threadIdx.x;

	for (int section = blockIdx.x;
		section < total_edge_sections;
		section += blocks
		) {

		// for each section

		double sum = 0;
		int v = 0;

		for (int j = idx + cu_edge_sections[section];
			j < cu_edge_sections[section + 1]; 
			j += blockSize) {

			v = flat_edges[j];

			sum += arr_pr[v + current * V] / arr_out_degree[v];

		}

		__shared__ double r[blockSize];
		r[idx] = sum;
		__syncthreads();
		for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
			if (idx < size)
				r[idx] += r[idx + size];
			__syncthreads();
		}

		if (idx == 0) {
			sections_result[section] = r[0];
			
		}

	}
}


__global__ void reduce_sections(
	const int V,
	const double d,
	const int next,
	const int* vertex_section_starts,
	double* arr_pr,
	const double* sections_result
	) {

	int idx = threadIdx.x;


	for (int vertexblock = blockIdx.x;
		vertexblock < V;
		vertexblock += blocks
		) {

		// for each vertexblock

		double sum = 0;

		for (int j = idx + vertex_section_starts[vertexblock];
			j < vertex_section_starts[vertexblock + 1]; 
			j += blockSize) {
			
			sum += sections_result[j];

		}

		__shared__ double r[blockSize];
		r[idx] = sum;
		__syncthreads();
		for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
			if (idx < size)
				r[idx] += r[idx + size];
			__syncthreads();
		}

		if (idx == 0) {
			arr_pr[vertexblock + next * V] = (1.0 - d) / V + d * r[0];

		}

	}
}



__global__ void allVertex(
	const int V,
	const double d,
	const int next,
	const int current,
	const int* flat_edges,
	const int* edge_starts,
	const int* arr_out_degree,
	double* arr_pr
) {

	int idx = threadIdx.x;
	

	for (int vertexblock = blockIdx.x;
		vertexblock < V;
		vertexblock += blocks
		) {

		// for each vertexblock

		double sum = 0;
		int v = 0; 

		for (int j = idx + edge_starts[vertexblock];
			j < edge_starts[vertexblock + 1]; j += blockSize) {
			v = flat_edges[j];

			sum += arr_pr[v + current * V] / arr_out_degree[v];

		}

		__shared__ double r[blockSize];
		r[idx] = sum;
		__syncthreads();
		for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
			if (idx < size)
				r[idx] += r[idx + size];
			__syncthreads();
		}

		if (idx == 0) {
			arr_pr[vertexblock + next * V] = (1.0 - d) / V + d * r[0];
		}

	}



}


__global__ void oneVertex(int i, 
	const int V, 
	const double d,
	const int next,
	const int current, 
	const int* flat_edges,
	const int* edge_starts,
	const int* arr_out_degree, 
	double* arr_pr) {

	int idx = threadIdx.x;
	double sum = 0;
	int v = 0;

	for (int j = idx + edge_starts[i];
		j < edge_starts[i + 1]; j += blockSize) {
		v = flat_edges[j];

		sum += arr_pr[v + current * V] / arr_out_degree[v];

	}

	__shared__ double r[blockSize];
	r[idx] = sum;
	__syncthreads();
	for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
		if (idx < size)
			r[idx] += r[idx + size];
		__syncthreads();
	}


	if (idx == 0) {
	
		arr_pr[i + next * V] = (1.0 - d) / V + d * r[0];
	}
	

	}


int main(int argc, char** argv) {
	FILE* fin = fopen(argv[1], "r");
	FILE* fout = fopen(argv[2], "w");
	fscanf(fin, "%d%d%d%d", &V, &E, &L, &M);
	in_edges.resize(V);
	out_degree = std::vector<int>(V, 0);

	int longest_in_edges = 0;
	for (int i = 0; i < E; ++i) {
		int u, v;
		fscanf(fin, "%d%d", &u, &v);
		in_edges[v].push_back(u);
		++out_degree[u];

		// compute longest edge
		if (in_edges[v].size() > longest_in_edges) {
			longest_in_edges = in_edges[v].size();
		}

	}



	std::vector<double> pr[2];
	pr[0].resize(V);
	pr[1].resize(V);
	int current = 0;
	for (int i = 0; i < V; ++i) {
		pr[current][i] = 1.0 / V;
	}

	//create array equivalents 

	int* flat_edges = (int*)malloc(E * sizeof(int));
	int* edge_starts = new int[V + 1];
	int* arr_out_degree = new int[V];
	double* arr_pr  = new double[V*2];
	//double arr_pr[V * 2];

	////cuda allocate PR 
	hipMallocManaged(&flat_edges, E * sizeof(int));
	hipMallocManaged(&edge_starts, (V + 1) * sizeof(int));
	hipMallocManaged(&arr_out_degree, V * sizeof(int));
	hipMallocManaged(&arr_pr, 2 * V * sizeof(double));


	//assign 
	int pos = 0;
	int total_edge_sections = 0; 

	int* edge_sections = (int*)malloc(500000 * sizeof(int));
	int* edge_section_to_vertex = (int*)malloc(500000 * sizeof(int));
	int* vertex_section_starts = new int[V + 1];

	for (int i = 0; i < V; ++i) {

		edge_starts[i] = pos;
		vertex_section_starts[i] = total_edge_sections;

		edge_sections[total_edge_sections] = pos; 
		edge_section_to_vertex[total_edge_sections] = i;

		++total_edge_sections;


		for (int j = 0; j < in_edges[i].size(); j++) {
			flat_edges[pos] = in_edges[i][j];
			
			++pos;


			if (((j + 1) % blockSize) == 0) {
				edge_sections[total_edge_sections] = pos;
				edge_section_to_vertex[total_edge_sections] = i;
				++total_edge_sections;
			}

		}

	}

	edge_starts[V] = E;
	edge_sections[total_edge_sections] = E;
	vertex_section_starts[V] = total_edge_sections; 

	int* cu_edge_sections = (int*)malloc((total_edge_sections+1) * sizeof(int));
	int* cu_edge_section_to_vertex = (int*)malloc(total_edge_sections * sizeof(int));
	double* sections_result = (double*)malloc(total_edge_sections * sizeof(double));
	int* cu_vertex_section_starts = new int[V + 1];

	hipMallocManaged(&cu_edge_sections, (total_edge_sections + 1) * sizeof(int));
	hipMallocManaged(&cu_edge_section_to_vertex, total_edge_sections * sizeof(int));
	hipMallocManaged(&sections_result, total_edge_sections * sizeof(double));
	hipMallocManaged(&cu_vertex_section_starts, (V + 1) * sizeof(int));



	for (int i = 0; i < total_edge_sections; ++i) {
		cu_edge_sections[i] = edge_sections[i];
		cu_edge_section_to_vertex[i] = edge_section_to_vertex[i];


		//cout << cu_edge_sections[i];
		//cout << edge_starts[i];
		//cout << endl;
	}
	cu_edge_sections[total_edge_sections] = E;

	//cout << total_edge_sections;
	//cout << endl;

	for (int i = 0; i < V + 1; ++i) {
		cu_vertex_section_starts[i] = vertex_section_starts[i];
	}


	for (int i = 0; i < V; ++i) {
		arr_out_degree[i] = out_degree[i];
	}

	for (int i = 0; i < V; ++i) {
		arr_pr[i + current * V] = 1.0 / V;
	}

	//// standard
	//for (int iter = 0; iter < M; ++iter) {
	//	int next = 1 - current;

	//	allVertex << <blocks, blockSize >> >(
	//	V,
	//	d,
	//	next,
	//	current,
	//	flat_edges,
	//	edge_starts,
	//	arr_out_degree,
	//	arr_pr
	//	);

	//	hipDeviceSynchronize();

	//	int same = 1;
	//	for (int i = 0; i < V; ++i) {
	//		if (arr_pr[i + current * V] != arr_pr[i + next * V]) {
	//			same = 0;
	//		}
	//	}

	//	if (same == 1) {
	//		break;
	//	}

	//	current = next;
	//}


	//// super cuda 



	for (int iter = 0; iter < M; ++iter) {
		int next = 1 - current;

		sum_sections << <blocks, blockSize >> >(
		V,
		total_edge_sections,
		current,
		flat_edges,
		cu_edge_sections,
		cu_edge_section_to_vertex,
		arr_out_degree,
		arr_pr,
		sections_result
		);

		//for (int i = 0; i < total_edge_sections; ++i) {
		//	cout << sections_result[i];
		//	cout << endl;
		//}

		//reduce_sections << <blocks, blockSize >> > (
		//	V,
		//	d,
		//	next,
		//	cu_vertex_section_starts,
		//	arr_pr,
		//	sections_result
		//	);

		//hipDeviceSynchronize();

		current = next;
	}


	// end stuff 

	hipDeviceSynchronize();

	for (int i = 0; i < V * 2; ++i) {
		//cout << arr_pr[i];
		//cout << edge_starts[i];

		//cout << endl;
	}


	cout << endl;

	for (int i = 0; i < V; ++i) {
		pr[current][i] = arr_pr[i + current * V];

	/*	cout << arr_pr[i + current * V];
		cout << endl;*/
	}

	for (int i = 0; i < V; ++i) {
		fprintf(fout, "%.8f\n", pr[current][i]);
	}

	hipFree(flat_edges);
	hipFree(edge_starts);
	hipFree(arr_out_degree);
	hipFree(arr_pr);
	hipFree(cu_edge_sections);
	hipFree(cu_edge_section_to_vertex);
	hipFree(sections_result);
	hipFree(cu_vertex_section_starts);


	fclose(fin);
	fclose(fout);

	return 0;
}


